#include <malloc.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <string.h>

#define M 256
#define N 256
#define Delta 0.001

#define BlocksX 16
#define ThreadsX 8
#define BlocksY 16
#define ThreadsY 8

#define U 100
#define D 100
#define L 0
#define R 0

void initialize(float *C);
void write2File(float *C, char name[]);
__global__ void redUpdate(float *C_gpu, float *e_gpu, int nX, int nY);
__global__ void blackUpdate(float *C_gpu, float *e_gpu, int nX, int nY);

/*
this function will find row-wise maximun in interval of size stride
maximum of this interval is stored in its first position
*/
__global__ void rowWiseReduction(float *e_gpu, int stride);
/*
this function will find column-wise maximun in interval of size stride
maximum of this interval is stored in its first position
*/
__global__ void columnWiseReduction(float *e_gpu, int stride);


int main(int argc, char *argv[]) {
	//Initialize C in cpu
	float *C;
	C = (float *)malloc((M * N) * sizeof(float));
	initialize(C);
	write2File(C, "00000.txt");

	//Send C to gpu
	float *C_gpu;
	hipMalloc((void**) &C_gpu, (M*N)*sizeof(float));
	hipMemcpy(C_gpu, C, (M*N)*sizeof(float), hipMemcpyHostToDevice);
	
	//define size of grids and blocks
	dim3 dimBlock(ThreadsX, ThreadsY);
	dim3 dimGrid(BlocksX, BlocksY);
	
  /*
	stopping variable delta
  delta_gpu is 2D array, stores local maximum changing of threads
  delta_gpu(i, j) is local maximum of thread (i, j)
  */
	float *delta_gpu, delta = 1;
	hipMalloc((void**) &delta_gpu, (BlocksX*BlocksY*ThreadsX*ThreadsY)*sizeof(float));
	
	int nX = M / (BlocksX*ThreadsX);
	int nY = N / (BlocksY*ThreadsY);

 	int stride;

	int k = 1;
  	char *iter_name;
  	iter_name = (char *)malloc(20*sizeof(char));

	do {
    //update red point in gpu
		redUpdate<<<dimGrid, dimBlock>>>(C_gpu, delta_gpu, nX, nY);

    //update black points in gpu
		blackUpdate<<<dimGrid, dimBlock>>>(C_gpu, delta_gpu, nX, nY);

    /*
    find row-wise max changing, store in postion (i, 0) of e_gpu
    stride is number of consecutive points in parallel reduction, doubled each time
    function rowWiseReduction is called until stride == dimension length
    */
    stride = 2;
    do {
      rowWiseReduction<<<dimGrid, dimBlock>>>(delta_gpu, stride);
      stride *= 2;
    } while (stride <= BlocksX*ThreadsX);

    /*
    find gloabal max changing by applying column-wise reduction
    global max is stored in position (0, 0) of e_gpu
    stride is number of consecutive points in parallel reduction, doubled each time
    function columnWiseReduction is called until stride == dimension length
    */
    stride = 2;
    do {
      columnWiseReduction<<<dimGrid, dimBlock>>>(delta_gpu, stride);
      stride *= 2;
    } while (stride <= BlocksY*ThreadsY);

    //send max changing to CPU
    hipMemcpy(&delta, delta_gpu, sizeof(float), hipMemcpyDeviceToHost);
    
    //save intermidiate states every 100 iterrations
    if (k % 100 == 0) {
      hipMemcpy(C, C_gpu, (M*N)*sizeof(float), hipMemcpyDeviceToHost);

      iter_name = (char *)malloc(50*sizeof(char));
      sprintf(iter_name, "%05d", k);
      strcat(iter_name, ".txt");

      write2File(C, iter_name);
    } 
    k += 1;
	} while (delta > Delta);
	
	//send final state to CPU
	hipMemcpy(C, C_gpu, (M*N)*sizeof(float), hipMemcpyDeviceToHost);
	hipFree(C_gpu);
	
	//save final result
	iter_name = (char *)malloc(50*sizeof(char));
	sprintf(iter_name, "%05d", k);
	strcat(iter_name, ".txt");
	write2File(C, iter_name);
}



void initialize(float *C)
{
    int i, j;
    for (i = 0; i < M; i++)
        for (j = 0; j < N; j++)
        {
            if (i >= (M / 2 - 20) && i < (M / 2 + 20) && j >= (N / 2 - 20) && j < (N / 2 + 20))
                *(C + i * N + j) = 80.0;
            else
                *(C + i * N + j) = 25.0;
        }
}

void write2File(float *C, char name[])
{
    char path[50] = "./results/CUDA/";
    strcat(path, name);

    FILE *result = fopen(path, "w");
    int i, j;

    for (i = 0; i < M; i++)
    {
        for (j = 0; j < N; j++)
        {
            fprintf(result, "%lf\t", *(C + i * N + j));
        }
        fprintf(result, "\n");
    }

    fclose(result);
}

__global__ void redUpdate(float *C_gpu, float *delta_gpu, int nX, int nY) {
	int index_x, start_x, end_x;
	start_x = (blockIdx.x * blockDim.x + threadIdx.x) * nX;
	end_x = (blockIdx.x * blockDim.x + threadIdx.x + 1) * nX;
		
	int index_y, start_y, end_y;
	start_y = (blockIdx.y * blockDim.y + threadIdx.y) * nY;
	end_y = (blockIdx.y * blockDim.y + threadIdx.y + 1) * nY;
	
	float u, d, l, r;
	float local_delta = 0;
	
	for (index_x = start_x; index_x < end_x; index_x ++ ) {
		for (index_y = start_y + ((index_x + start_y) % 2); index_y < end_y; index_y += 2) {
		
			//check boundary condition dim x
			if (index_x == 0) {
				u = U;
				d = *(C_gpu + (index_x + 1) * N + index_y);
			} else if (index_x == M - 1){
				u = *(C_gpu + (index_x - 1) * N + index_y);
				d = D;	
			} else {
				u = *(C_gpu + (index_x-1) * N + index_y);
				d = *(C_gpu + (index_x+1) * N + index_y);
			}
			
			//check boundary condition dim y
			if (index_y == 0) {
				l = L;
				r = *(C_gpu + index_x * N + index_y + 1);
			} else if (index_y == N - 1) {
				l = *(C_gpu + index_x * N + index_y - 1);
				r = R;
			} else {
				l = *(C_gpu + index_x * N + index_y - 1);
				r = *(C_gpu + index_x * N + index_y + 1);
			}
			
			//update local delta
			if (abs(*(C_gpu + index_x * N + index_y) - (u + d + l + r)/4) > local_delta) {
				local_delta = abs(*(C_gpu + index_x * N + index_y) - (u + d + l + r)/4);
			}
			
			//update point value
			*(C_gpu + index_x * N + index_y) = (u + d + l + r)/4;		
		}
	}
	*(delta_gpu + (blockIdx.x*blockDim.x + threadIdx.x)*blockDim.y*BlocksY + blockIdx.y*blockDim.y + threadIdx.y) = local_delta;
}


__global__ void blackUpdate(float *C_gpu, float *delta_gpu, int nX, int nY) {
	int index_x, start_x, end_x;
	start_x = (blockIdx.x * blockDim.x + threadIdx.x) * nX;
	end_x = (blockIdx.x * blockDim.x + threadIdx.x + 1) * nX;
		
	int index_y, start_y, end_y;
	start_y = (blockIdx.y * blockDim.y + threadIdx.y) * nY;
	end_y = (blockIdx.y * blockDim.y + threadIdx.y + 1) * nY;
	
	float u, d, l, r;
	float local_delta = 0;
	
	for (index_x = start_x; index_x < end_x; index_x ++ ) {
		for (index_y = start_y + ((index_x + start_y + 1) % 2); index_y < end_y; index_y += 2) {
		
			//check boundary condition dim x
			if (index_x == 0) {
				u = U;
				d = *(C_gpu + (index_x + 1) * N + index_y);
			} else if (index_x == M - 1){
				u = *(C_gpu + (index_x - 1) * N + index_y);
				d = D;	
			} else {
				u = *(C_gpu + (index_x-1) * N + index_y);
				d = *(C_gpu + (index_x+1) * N + index_y);
			}
			
			//check boundary condition dim y
			if (index_y == 0) {
				l = L;
				r = *(C_gpu + index_x * N + index_y + 1);
			} else if (index_y == N - 1) {
				l = *(C_gpu + index_x * N + index_y - 1);
				r = R;
			} else {
				l = *(C_gpu + index_x * N + index_y - 1);
				r = *(C_gpu + index_x * N + index_y + 1);
			}
			
			//update local delta
			if (abs(*(C_gpu + index_x * N + index_y) - (u + d + l + r)/4) > local_delta) {
				local_delta = abs(*(C_gpu + index_x * N + index_y) - (u + d + l + r)/4);
			}
			
			//update point value
			*(C_gpu + index_x * N + index_y) = (u + d + l + r)/4;	
		}
	}
	if (*(delta_gpu + (blockIdx.x*blockDim.x + threadIdx.x)*blockDim.y + threadIdx.y) < local_delta) {
	  *(delta_gpu + (blockIdx.x*blockDim.x + threadIdx.x)*blockDim.y*BlocksY + blockIdx.y*blockDim.y + threadIdx.y) = local_delta;
	}
}

/*
this function will find row-wise maximun in interval of size stride
maximum of this interval is stored in its first position
*/
__global__ void rowWiseReduction(float *delta_gpu, int stride) {
  int index_y = blockIdx.y*blockDim.y + threadIdx.y;
  int index_x = blockIdx.x*blockDim.x + threadIdx.x;

  if (index_y % stride == 0) {
    if (*(delta_gpu + index_x*blockDim.y*BlocksY + index_y) < *(delta_gpu + index_x*blockDim.y*BlocksY + index_y + stride/2)) {
      *(delta_gpu + index_x*blockDim.y*BlocksY + index_y) = *(delta_gpu + index_x*blockDim.y*BlocksY + index_y + stride/2);
    }
  }
}


/*
this function will find column-wise maximun in interval of size stride
maximum of this interval is stored in its first position
*/
__global__ void columnWiseReduction(float *delta_gpu, int stride) {
  int index_y = blockIdx.y*blockDim.y + threadIdx.y;
  int index_x = blockIdx.x*blockDim.x + threadIdx.x;

  if ((index_x % stride == 0) && (index_y == 0)) {
    if (*(delta_gpu + index_x*blockDim.y*BlocksY + index_y) < *(delta_gpu + (index_x + stride / 2)*blockDim.y*BlocksY + index_y)) {
      *(delta_gpu + index_x*blockDim.y*BlocksY + index_y) = *(delta_gpu + (index_x + stride / 2)*blockDim.y*BlocksY + index_y);
    }
  }
}
